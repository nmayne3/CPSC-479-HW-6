#include "hip/hip_runtime.h"
#include <stdio.h>

// Function for printing arrays
__host__ void print_matrix(int *matrix, int N) {
	printf("N = %d\n", N);
	for (int i = 0; i<N; ++i) {
		for(int j = 0; j < N; ++j) {
			printf("%d, ", matrix[i*N + j]);
		}
		printf("\n");
	}
}

// Populates matrix with random values between 0 -- 10
__host__ void populate(int* matrix, int N) {
	srand(time(NULL));
	int count = 0;
	for(int i = 0; i < N; ++i) {
		for(int j = 0; j < N; j++) {
			matrix[i*N + j] = rand() % 10; 
			if(matrix[i*N + j] % 2 == 0)
				count++;
		}
	}
}

__global__ void count(int *matrix, int *count, int N) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < N*N) {
		if(matrix[id] % 2 == 0) {
			atomicAdd(count, 1);
		}
	}
}

__global__ void square(int *matrix, int *result, int N) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned i = id / N;
	unsigned j = id % N;
	for (unsigned k = 0; k < N; ++k) {
		result[i*N + j] += matrix[i*N + k] *
			matrix[k*N + j];
	}
}


int main() {
	// Exercise 1
	int N = 16;

	// Create matrices of size N
	int *h_inmatrix = (int*)malloc(N*N*sizeof(int));
	int *hcount = (int*)malloc(sizeof(int));
	int *d_inmatrix;
	int *d_outmatrix;
	int *dcount;
	hipMalloc((void **)&d_inmatrix, N*N*sizeof(int));
	hipMalloc((void **)&d_outmatrix, N*N*sizeof(int));
	hipMalloc((void **)&dcount, sizeof(int));

	populate(h_inmatrix, N);
	hipMemcpy(d_inmatrix, h_inmatrix, N*N*sizeof(int), hipMemcpyHostToDevice);
	print_matrix(h_inmatrix, N);

	*hcount = 0;
	hipMemcpy(dcount, hcount, sizeof(int), hipMemcpyHostToDevice);
	count<<<N, N>>>(d_inmatrix, dcount, N);
	hipMemcpy(hcount, dcount, sizeof(int), hipMemcpyDeviceToHost);
	printf("Number of Even values: %d\n\n", *hcount);

	square<<<N, N>>>(d_inmatrix, d_outmatrix, N);
	hipMemcpy(h_inmatrix, d_outmatrix, N*N*sizeof(int), hipMemcpyDeviceToHost);
	print_matrix(h_inmatrix, N);

	hipFree(d_inmatrix);
	hipFree(d_outmatrix);
	hipFree(dcount);
	free(h_inmatrix);

	return 0;
}
